#include "hip/hip_runtime.h"
#include "applyIterGPU.h"

__global__
void GPU_PAR_FOR_HELPER(int height, int width,double* values, double* zr, double* zi, double* cr, double* ci, size_t max_iter)
{
  double iters=0,
         R2=1e6,
         zr2=0,
         zi2=0,
         q;
   
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  
  if (i<height*width)
  {
    // q is used to determine if a point is within the set
    // without needing to iterate to max_iter
    q = (cr[i]-1./4)*(cr[i]-1./4) + ci[i]*ci[i];

    if (q*(q+(cr[i]-1./4)) <= 1./4*ci[i]*ci[i])
      iters=max_iter;
    else if ((cr[i]+1)*(cr[i]+1) + ci[i]*ci[i] <= 1./16)
      iters=max_iter;
    while((zr2+zi2<=R2) && (iters<max_iter))
    {
      zi[i] = zi[i] * zr[i];
      zi[i] = zi[i] + zi[i] + ci[i];
      zr[i] = zr2 - zi2 + cr[i];
      zr2 = zr[i]* zr[i];
      zi2 = zi[i]* zi[i]; 
      iters++;
    }
    values[i] = iters;
  }
}

void applyIterGPU::GPU_PAR_FOR(int height, int width)
{
  double *GPUvalues, *GPUzr, *GPUzi, *GPUcr, *GPUci;
  
  hipMalloc(&GPUvalues, height*width*sizeof(double));
  hipMemcpy(GPUvalues, values, height*width*sizeof(double),hipMemcpyHostToDevice);
  
  hipMalloc(&GPUzr, height*width*sizeof(double));
  hipMemcpy(GPUzr, zr, height*width*sizeof(double),hipMemcpyHostToDevice);
  
  hipMalloc(&GPUzi, height*width*sizeof(double));
  hipMemcpy(GPUzi, zi, height*width*sizeof(double),hipMemcpyHostToDevice);
  
  hipMalloc(&GPUcr, height*width*sizeof(double));
  hipMemcpy(GPUcr, cr, height*width*sizeof(double),hipMemcpyHostToDevice);
  
  hipMalloc(&GPUci, height*width*sizeof(double));
  hipMemcpy(GPUci, ci, height*width*sizeof(double),hipMemcpyHostToDevice);

  GPU_PAR_FOR_HELPER<<<((height*width)+255)/256, 256>>>(height, width, GPUvalues, GPUzr, GPUzi, GPUcr, GPUci, max_iter);

  hipMemcpy(values, GPUvalues, height*width*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(zr, GPUzr, height*width*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(zi, GPUzi, height*width*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(cr, GPUcr, height*width*sizeof(double),hipMemcpyDeviceToHost);
  hipMemcpy(ci, GPUci, height*width*sizeof(double),hipMemcpyDeviceToHost);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  hipFree(GPUvalues);
  hipFree(GPUzr);
  hipFree(GPUzi);
  hipFree(GPUcr);
  hipFree(GPUci);
}

