#include "hip/hip_runtime.h"
#include "applyIterGPU.h"
#include "multi_prec/multi_prec_certif.h"

__device__
void calcPoint(multi_prec<2>& cr, multi_prec<2>& ci, multi_prec<2>& centerx, multi_prec<2>& centery, multi_prec<2>& zoom, int& width, int& height, float& i, float& j)
{
  float aspect_ratio = float(width)/height;
  multi_prec<2> x_range, y_range, xmin, ymin, intervalx, intervaly;
  if (aspect_ratio<1){
    x_range = 4/zoom;
    y_range = (1/aspect_ratio)*4/zoom;
  }
  else{
    x_range = (aspect_ratio)*4/zoom;
    y_range = 4/zoom;
  }
  xmin = centerx - x_range/2;
  ymin = centery - y_range/2;
  intervalx = x_range/width;
  intervaly = y_range/height;
  
  cr = xmin + i*intervalx;
  ci = ymin + j*intervaly;
}

__global__
void GPU_PAR_FOR_HELPER(int height, int width,float* values, multi_prec<2> centerx, multi_prec<2> centery, multi_prec<2> zoom, size_t max_iter)
{
  multi_prec<2> R2=10.,
         zr=0.,
         zi=0.,
         cr=0.,
         ci=0.,
         zr2=0.,
         zi2=0.,
         q;

  float iters = 0;
   
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  
  if (idx < height*width)
  {
    iters=0.;
    zr=0.;
    zi=0.;
    cr=0.;
    ci=0.;
    zr2=0.;
    zi2=0.;
    
    float i = idx/height;
    float j = idx - height*i;
    calcPoint(cr,ci,centerx,centery,zoom,width,height,i,j);
    
    // q is used to determine if a point is within the set
    // without needing to iterate to max_iter
    q = (cr-1./4)*(cr-1./4) + ci*ci;

    if (q*(q+(cr-1./4)) <= 1./4*ci*ci)
      iters=max_iter;
    else if ((cr+1)*(cr+1) + ci*ci <= 1./16)
      iters=max_iter;
    
    while((zr2+zi2<=R2) && (iters<max_iter))
    {
      zi = zi * zr;
      zi = zi + zi + ci;
      zr = zr2 - zi2 + cr;
      zr2 = zr* zr;
      zi2 = zi* zi; 
      iters++;
    }
    values[idx] = iters;
  }
}

applyIterGPU::applyIterGPU(int height, int width, size_t max_iter): 
  height(height),
  width(width),
  max_iter(max_iter)
{
  hipMalloc(&values, height*width*sizeof(float));
  /*hipMalloc(&zr, height*width*sizeof(float));
  hipMalloc(&zi, height*width*sizeof(float));
  hipMalloc(&cr, height*width*sizeof(float));
  hipMalloc(&ci, height*width*sizeof(float));*/
}

applyIterGPU::~applyIterGPU()
{
  hipFree(values);
  /*hipFree(zr);
  hipFree(zi);
  hipFree(cr);
  hipFree(ci);*/
}

void applyIterGPU::SET_COORD_VALS(std::string centerx, std::string centery, std::string zoom)
{
  this->centerx = centerx;
  this->centery = centery;
  this->zoom = zoom;
  //SET_COORD_VALS_HELPER<<<(width+127)/128, 128>>>(zr, zi, cr, ci, centerx, centery, zoom, width, height);
}
    
  
void applyIterGPU::GPU_PAR_FOR()
{ 
  multi_prec<2> centerx_ = centerx.c_str(),
                centery_ = centery.c_str(),
                zoom_ = zoom.c_str(); 

  GPU_PAR_FOR_HELPER<<<(height*width+255)/256, 256>>>(height, width, values, centerx_, centery_, zoom_, max_iter);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
}

void applyIterGPU::copyValues(float* target)
{
  hipMemcpy(target, values, height*width*sizeof(float),hipMemcpyDeviceToHost);
}

